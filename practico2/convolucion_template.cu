#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <Windows.h>
#include "util.h"
#include <time.h>
//#include <sys\time.h>

#define CHUNK 16
#define SIZE_X 2048
#define MASK_SIZE 3


void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }
}

// declarar máscara en memoria constante...
// ...

__global__ void Kernel_Convolucion(int * inputArray, int* outputArray, int* mask, int arraySize, int maskSize){

	int i = 0,
		j = 0,
		position,
		radio = (int)maskSize / 2;

	for (i = - radio; i < (threadIdx.x + radio); i++) {
		if (i >= 0 && i < arraySize ) {                    
			if (blockIdx.x == 1 && threadIdx.x == 1) {
				printf("El primer hilo suma en el indice [%d * %d + %d] * mask[%d + %d]",blockDim.x,blockIdx.x,threadIdx.x,i,i,maskSize);
			}
			outputArray[blockDim.x * blockIdx.x  + threadIdx.x] += inputArray[blockDim.x * blockIdx.x  + threadIdx.x + i] * mask[i + maskSize];
		}
	}

}

__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray){

}

__global__ void Kernel_Convolucion_Shared(int * inputArray, int* outputArray, int* mask){

}

void Convolucion_C(int * inputArray, int* ouputArray, int * mask)
{
	int i, j;

	for( i = 0; i<SIZE_X;i++){   
		ouputArray[i] = 0;
		for( j =0; j<MASK_SIZE;j++){      
			int position = i-(int)(MASK_SIZE/2) + j;
			if(position>=0 && position<SIZE_X)
				ouputArray[i] += inputArray[position] * mask[j];
		}       
	}
} 

int main() {

	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_CPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE);

	int i;

	struct timeval a, b,c,d,e;

	// arrays en el device
	int * inputArray_dev;
	int * outputArray_dev;	
	int* mask_dev;	

	float t_i, t_f, t_sys, diff;

	// memoria para arrays en dispositivo
	inputArray_dev = (int*)malloc(sizeof(int) * SIZE_X);
	outputArray_dev = (int*)malloc(sizeof(int) * SIZE_X);
	mask_dev = (int*)malloc(sizeof(int) * MASK_SIZE);


	cudaCheck();

	for (i = 0; i < SIZE_X; i++){
		inputArray[i] = i % 10;
		inputArray_dev[i] = i % 10;
		outputArray_CPU[i] = 0;
		outputArray_GPU[i] = 0;
		//	outputArray_dev[i] = 0; // ????
	}		

	//definir una máscara...
	for (i = 0; i < MASK_SIZE; i++) {
		mask[i] = 1;
		//	mask_dev[i] = 1; // ?????
	}

	// Convolución en CPU...	
	clockStart();
	Convolucion_C(inputArray, outputArray_CPU, mask);
	clockStop("CPU");

	// copiar array de entrada al dispositivo...
	hipMemcpy(inputArray_dev, inputArray, SIZE_X, hipMemcpyHostToDevice);
	hipMemcpy(outputArray_dev, outputArray_GPU, SIZE_X, hipMemcpyHostToDevice);
	hipMemcpy(mask_dev, mask, MASK_SIZE, hipMemcpyHostToDevice);

	// setear en 0 el array de salida en el dispositivo...
	// ...
	//hipMemset(outputArray_dev, 0, SIZE_X);
		

	// copiar la máscara o setear la máscara en memoria constante (hipMemcpyToSymbol)
	// ...

	int cantBloques = SIZE_X / CHUNK;
	int tamBloque = CHUNK;

	clockStart();
	Kernel_Convolucion<<<cantBloques, tamBloque>>>(inputArray_dev, outputArray_dev, mask_dev, SIZE_X, MASK_SIZE);
	hipDeviceSynchronize();
	clockStop("GPU");
 
	// copiar array de salida desde el dispositivo...
	hipMemcpy(outputArray_GPU,outputArray_dev,SIZE_X,hipMemcpyDeviceToHost);

	// chequear salida...
	for(i = 0; i < SIZE_X; i++){
		printf("Joaco se la : %d ::: %d \n", outputArray_CPU[i],  outputArray_GPU[i]);
		
		if (outputArray_CPU[i] != outputArray_GPU[i]){
			printf("outputArray_CPU[%d] != outputArray_GPU[%d] \n",i,i);
			break;
		}
	}	

	// liberar memoria cpu...
	free(inputArray);
	free(outputArray_CPU);
	free(outputArray_GPU);
	free(mask);

	// liberar memoria dispositivo...
	hipFree(inputArray_dev);
	hipFree(outputArray_dev);
	//hipFree(outputArray_GPU);
	hipFree(mask_dev);

	return 0;
}
