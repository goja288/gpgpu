#include "hip/hip_runtime.h"
/*******************/
/** Ejercicio 3 a **/
/*******************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <Windows.h>
#include "util.h"
#include <time.h>
//#include <sys\time.h>

#define CHUNK 256
#define SIZE_X 1048576
#define MASK_SIZE 21

void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }
}

// declarar m�scara en memoria constante...
// ...

__global__ void Kernel_Convolucion(int * inputArray, int* outputArray, int* mask){

}

__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray){

}

__global__ void Kernel_Convolucion_Shared(int * inputArray, int* outputArray, int* mask){
	
	__shared__ int elements[ CHUNK + 2*(int)(MASK_SIZE/2) ];

	int start	= blockDim.x * blockIdx.x  + threadIdx.x,
		radio	= (int)(MASK_SIZE / 2),
		i		= 0,
		j		= 0,
		maskInd = 0;

	elements[threadIdx.x + radio] = inputArray[start];

	if (threadIdx.x == 0 ) {
		
		if (start == 0 ) {
			for (i = 0 ; i < radio; i++) {
				elements[i] = 0 ;	
			}
			
		} else {
			for(i = 0 ; i < radio; i++){
				elements[ i ] = inputArray[start - radio + i ];
			}
		}
		
	} 
	else if (threadIdx.x == CHUNK - 1) {
		
		if (start == SIZE_X - 1 ){ // Soy el ultimo elemento del array de entrada?
			for (i = 0; i < radio; i++){
				elements[CHUNK + radio + i ] = 0;
			}
		} else { 	
			// Soy el ultimo elemento del bloque
			for (i = 0; i < radio; i++) {
				elements[blockDim.x + radio + i ] = inputArray[start + i + 1];
			}
		}
		
	}
	__syncthreads(); 
	
	
	
	int centro = threadIdx.x + radio,
		min = centro - radio,
		max = centro + radio,
		ac = 0	;
	
	for(i = threadIdx.x ; i <= threadIdx.x + 2*radio; i++){
		
		ac+= elements[ i ] *mask[maskInd]  ;
		//printf(" %d \n" , maskInd);
		maskInd++;
		
	}

	outputArray[start] = ac;
}

void Convolucion_C(int * inputArray, int* ouputArray, int * mask)
{
	int i, j;

	for( i = 0; i<SIZE_X;i++){   
		ouputArray[i] = 0;
		for( j =0; j<MASK_SIZE;j++){      
			int position = i-(int)(MASK_SIZE/2) + j;
			if(position>=0 && position<SIZE_X)
				ouputArray[i] += inputArray[position] * mask[j];
		}       
	}
} 

int main() {

	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_CPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE); 
	int* mask_dev;
	int i;

	struct timeval a, b,c,d,e;

	// arrays en el device
	int * inputArray_dev;
	int * outputArray_dev;	
	//int* mask_dev;	

	float t_i, t_f, t_sys, diff;

	// memoria para arrays en dispositivo
	hipMalloc(&inputArray_dev, sizeof(int)*SIZE_X);
	hipMalloc(&outputArray_dev, sizeof(int) * SIZE_X);
	hipMalloc(&mask_dev, sizeof(int) * MASK_SIZE);

	cudaCheck();

	for (i = 0; i < SIZE_X; i++){
		inputArray[i] = 1;
		outputArray_CPU[i] = 0;
		outputArray_GPU[i] = 0;
	}		

	//definir una m�scara...
	for (i = 0; i < MASK_SIZE; i++) {
		mask[i] = 1;
	}

	// Convoluci�n en CPU...	
	clockStart();
	Convolucion_C(inputArray, outputArray_CPU, mask);
	clockStop("CPU");

	// copiar array de entrada al dispositivo...
	hipMemcpy(inputArray_dev, inputArray,  sizeof(int) *SIZE_X, hipMemcpyHostToDevice);
	hipMemcpy(outputArray_dev, outputArray_GPU,  sizeof(int) *SIZE_X, hipMemcpyHostToDevice);
	hipMemcpy(mask_dev, mask, sizeof(int) * MASK_SIZE, hipMemcpyHostToDevice);

	int cantBloques = SIZE_X / CHUNK;
	int tamBloque = CHUNK;

	
	clockStart();
	Kernel_Convolucion_Shared<<<cantBloques, tamBloque>>>(inputArray_dev, outputArray_dev, mask_dev);
	hipDeviceSynchronize();
	clockStop("GPU");
	cudaCheck();
 
	// copiar array de salida desde el dispositivo...
	hipMemcpy(outputArray_GPU,outputArray_dev,sizeof(int) *SIZE_X,hipMemcpyDeviceToHost);

	// chequear salida...
	for(i = 0; i < SIZE_X; i++){
		if (outputArray_CPU[i] != outputArray_GPU[i]){
			printf("outputArray_CPU[%d] != outputArray_GPU[%d] \n",i,i);
			break;
		}
	}	

	// if (i == SIZE_X -1 )	printf("OK !!" );

	// liberar memoria cpu...
	free(inputArray);
	free(outputArray_CPU);
	free(outputArray_GPU);
	free(mask);

	// liberar memoria dispositivo...
	hipFree(inputArray_dev);
	hipFree(outputArray_dev);
	hipFree(mask_dev);

	return 0;
}

