#include "hip/hip_runtime.h"
/*******************/
/** Ejercicio 2 a **/
/*******************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <Windows.h>
#include "util.h"
#include <time.h>
//#include <sys\time.h>

#define CHUNK 256
#define SIZE_X 1048576
#define MASK_SIZE 21


// Declaramos la memoria constante
__constant__ int mask_dev[MASK_SIZE];

void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }
}

// declarar m�scara en memoria constante...
// ...

__global__ void Kernel_Convolucion(int * inputArray, int* outputArray, int* mask, int arraySize, int maskSize){

	
}

__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray){
	int start = blockDim.x * blockIdx.x  + threadIdx.x,
		i = 0,
		maskInd = 0,
		radio = (int)MASK_SIZE / 2;

	for (i =start - radio; i <= (start + radio); i++) {
		if (i >= 0 && i < SIZE_X ) {              
			outputArray[start] += inputArray[i] * mask_dev[maskInd] ;
		}
		maskInd++;
	}
}

__global__ void Kernel_Convolucion_Shared(int * inputArray, int* outputArray, int* mask){

}

void Convolucion_C(int * inputArray, int* ouputArray, int * mask)
{
	int i, j;

	for( i = 0; i<SIZE_X;i++){   
		ouputArray[i] = 0;
		for( j =0; j<MASK_SIZE;j++){      
			int position = i-(int)(MASK_SIZE/2) + j;
			if(position>=0 && position<SIZE_X)
				ouputArray[i] += inputArray[position] * mask[j];
		}       
	}
} 

int main() {

	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_CPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE); 

	int i;

	struct timeval a, b,c,d,e;

	// arrays en el device
	int * inputArray_dev;
	int * outputArray_dev;	
	//int* mask_dev;	

	float t_i, t_f, t_sys, diff;

	// memoria para arrays en dispositivo
	hipMalloc(&inputArray_dev, sizeof(int)*SIZE_X);
	hipMalloc(&outputArray_dev, sizeof(int) * SIZE_X);
//	hipMalloc(&mask_dev, sizeof(int) * MASK_SIZE);

	cudaCheck();

	for (i = 0; i < SIZE_X; i++){
		inputArray[i] = i % 10;
		outputArray_CPU[i] = 0;
		outputArray_GPU[i] = 0;
		//	outputArray_dev[i] = 0; // ????
	}		

	//definir una m�scara...
	for (i = 0; i < MASK_SIZE; i++) {
		mask[i] = 1;
		//	mask_dev[i] = 1; // ?????
	}

	// Convoluci�n en CPU...	
	clockStart();
	Convolucion_C(inputArray, outputArray_CPU, mask);
	clockStop("CPU");

	// copiar array de entrada al dispositivo...
	hipMemcpy(inputArray_dev, inputArray,  sizeof(int) *SIZE_X, hipMemcpyHostToDevice);
	hipMemcpy(outputArray_dev, outputArray_GPU,  sizeof(int) *SIZE_X, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(mask_dev), mask, sizeof(int) * MASK_SIZE);
	
	
	int cantBloques = SIZE_X / CHUNK;
	int tamBloque = CHUNK;

	clockStart();
	Kernel_Convolucion_Constante<<<cantBloques, tamBloque>>>(inputArray_dev, outputArray_dev);
	hipDeviceSynchronize();
	clockStop("GPU");cudaCheck();
 
	// copiar array de salida desde el dispositivo...
	hipMemcpy(outputArray_GPU,outputArray_dev,sizeof(int) *SIZE_X,hipMemcpyDeviceToHost);

	// chequear salida...
	for(i = 0; i < SIZE_X; i++){
				
		if (outputArray_CPU[i] != outputArray_GPU[i]){
			printf("outputArray_CPU[%d] != outputArray_GPU[%d] \n",i,i);
			break;
		}
	}	

	// liberar memoria cpu...
	free(inputArray);
	free(outputArray_CPU);
	free(outputArray_GPU);
	free(mask);

	// liberar memoria dispositivo...
	hipFree(inputArray_dev);
	hipFree(outputArray_dev);
	//hipFree(outputArray_GPU);
	//hipFree(mask_dev);

	return 0;
}

